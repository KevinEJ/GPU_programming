#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <cassert>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *gradient,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox )
{	
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
    
    const int up_n    = wt*(yt-1) + xt   ; 
    const int down_n  = wt*(yt+1) + xt   ; 
    const int left_n  = wt*yt     + xt-1 ; 
    const int right_n = wt*yt     + xt+1 ; 
    
    //if( mask[up_n] < 127.0f or mask[down_n] < 127.0f )
        bool b_up    = ( yt!=0  and mask[curt] > 127.0f and mask[up_n] > 127.0f)? false : true ; 
        bool b_down  = ( yt!=(ht-1) and mask[curt] > 127.0f and mask[down_n] > 127.0f)? false : true ; 
        bool b_left  = ( xt!=0  and mask[curt] > 127.0f and mask[left_n] > 127.0f)? false : true ; 
        bool b_right = ( xt!=(wt-1) and mask[curt] > 127.0f and mask[right_n] >127.0f)? false : true ; 
    
        const int yb = oy+yt, xb = ox+xt;
        const int curb_up    = wb*(yb-1) +xb  ;
        const int curb_down  = wb*(yb+1) +xb  ;
        const int curb_left  = wb*yb     +xb-1;
        const int curb_right = wb*yb     +xb+1;
    
    float Rt_up    = (b_up    )? 0 :target[up_n*3]     ; 
    float Rt_down  = (b_down  )? 0 :target[down_n*3]   ; 
    float Rt_left  = (b_left  )? 0 :target[left_n*3]   ; 
    float Rt_right = (b_right )? 0 :target[right_n*3]  ; 
    float Gt_up    = (b_up    )? 0 :target[up_n*3+1]   ; 
    float Gt_down  = (b_down  )? 0 :target[down_n*3+1] ; 
    float Gt_left  = (b_left  )? 0 :target[left_n*3+1] ; 
    float Gt_right = (b_right )? 0 :target[right_n*3+1]; 
    float Bt_up    = (b_up    )? 0 :target[up_n*3+2]   ; 
    float Bt_down  = (b_down  )? 0 :target[down_n*3+2] ; 
    float Bt_left  = (b_left  )? 0 :target[left_n*3+2] ; 
    float Bt_right = (b_right )? 0 :target[right_n*3+2]; 
        
    int n_of_b = 4 - ( (int)b_up + (int)b_down + (int)b_left + (int)b_right ) ;
    if( yt < ht and xt < wt and mask[curt]>127.0f){
        gradient[curt*3]   = ( n_of_b * target[curt*3]   - Rt_up - Rt_down - Rt_left - Rt_right) / n_of_b ;  
        gradient[curt*3+1] = ( n_of_b * target[curt*3+1] - Gt_up - Gt_down - Gt_left - Gt_right) / n_of_b ;  
        gradient[curt*3+2] = ( n_of_b * target[curt*3+2] - Bt_up - Bt_down - Bt_left - Bt_right) / n_of_b ;  
    //mask boundary condition
       /* 
        gradient[curt*3] += background[curb_up*3]*(int)(b_up) ;
        gradient[curt*3] += background[curb_down*3]*(int)(b_down) ;
        gradient[curt*3] += background[curb_left*3]*(int)(b_left) ;
        gradient[curt*3] += background[curb_right*3]*(int)(b_right) ;
        gradient[curt*3+1] += background[curb_up*3+1]*(int)(b_up) ;
        gradient[curt*3+1] += background[curb_down*3+1]*(int)(b_down) ;
        gradient[curt*3+1] += background[curb_left*3+1]*(int)(b_left) ;
        gradient[curt*3+1] += background[curb_right*3+1]*(int)(b_right) ;
        gradient[curt*3+2] += background[curb_up*3+2]*(int)(b_up) ;
        gradient[curt*3+2] += background[curb_down*3+2]*(int)(b_down) ;
        gradient[curt*3+2] += background[curb_left*3+2]*(int)(b_left) ;
        gradient[curt*3+2] += background[curb_right*3+2]*(int)(b_right) ;*/
    }else{
        gradient[curt*3] =   target[curt*3] ;
        gradient[curt*3+1] = target[curt*3+1] ;
        gradient[curt*3+2] = target[curt*3+2] ;
    }
}

__global__ void PoissonImageCloningIteration(
    const float *background,
	const float *fixed,
	const float *mask,
	const float *target,
	float *buf2,
	const int wb, const int hb, const int wt, const int ht,
    const int oy , const int ox)
{	
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
    
    const int up_n    = wt*(yt-1) + xt   ; 
    const int down_n  = wt*(yt+1) + xt   ; 
    const int left_n  = wt*yt     + xt-1 ; 
    const int right_n = wt*yt     + xt+1 ; 
    
    float Rt_up    = (yt==0)?  0 :target[up_n*3] ; 
    float Rt_down  = (yt==(ht-1))? 0 :target[down_n*3] ; 
    float Rt_left  = (xt==0)?  0 :target[left_n*3] ; 
    float Rt_right = (xt==(wt-1))? 0 :target[right_n*3] ; 
    float Gt_up    = (yt==0)?  0 :target[up_n*3+1] ; 
    float Gt_down  = (yt==(ht-1))? 0 :target[down_n*3+1] ; 
    float Gt_left  = (xt==0)?  0 :target[left_n*3+1] ; 
    float Gt_right = (xt==(wt-1))? 0 :target[right_n*3+1] ; 
    float Bt_up    = (yt==0)?  0 :target[up_n*3+2] ; 
    float Bt_down  = (yt==(ht-1))? 0 :target[down_n*3+2] ; 
    float Bt_left  = (xt==0)?  0 :target[left_n*3+2] ; 
    float Bt_right = (xt==(wt-1))? 0 :target[right_n*3+2] ; 
    //if( mask[up_n] < 127.0f or mask[down_n] < 127.0f )
        const int yb = oy+yt, xb = ox+xt;
        const int curb_up    = wb*(yb-1) +xb  ;
        const int curb_down  = wb*(yb+1) +xb  ;
        const int curb_left  = wb*yb     +xb-1;
        const int curb_right = wb*yb     +xb+1;

    if( yt < ht and xt < wt and mask[curt]>127.0f){
    //mask boundary condition
        bool b_up    = ( yt==0  or mask[up_n]<127.0f)? true : false ; 
        bool b_down  = ( yt==(ht-1) or mask[down_n]<127.0f)? true : false ; 
        bool b_left  = ( xt==0  or mask[left_n]<127.0f)? true : false ; 
        bool b_right = ( xt==(wt-1) or mask[right_n]<127.0f)? true : false ; 
        int n_of_b = 4 ; // - ( (int)b_up + (int)b_down + (int)b_left + (int)b_right ) ;
        buf2[curt*3+0] = ( 4*fixed[curt*3+0] + Rt_up*(int)(!b_up)      + background[3*curb_up]   *(int)(b_up)
                                           + Rt_down*(int)(!b_down)  + background[3*curb_down] *(int)(b_down)
                                           + Rt_left*(int)(!b_left)  + background[3*curb_left] *(int)(b_left)
                                           + Rt_right*(int)(!b_right)+ background[3*curb_right]*(int)(b_right) ) 
                        / (float)n_of_b ;  
        buf2[curt*3+1] = ( 4*fixed[curt*3+1] + Gt_up*(int)(!b_up)      + background[3*curb_up+1]   *(int)(b_up)
                                           + Gt_down*(int)(!b_down)  + background[3*curb_down+1] *(int)(b_down)
                                           + Gt_left*(int)(!b_left)  + background[3*curb_left+1] *(int)(b_left)
                                           + Gt_right*(int)(!b_right)+ background[3*curb_right+1]*(int)(b_right) ) 
                        / (float)n_of_b ;  
        buf2[curt*3+2] = ( 4*fixed[curt*3+2] + Bt_up*(int)(!b_up)      + background[3*curb_up+2]   *(int)(b_up)
                                           + Bt_down*(int)(!b_down)  + background[3*curb_down+2] *(int)(b_down)
                                           + Bt_left*(int)(!b_left)  + background[3*curb_left+2] *(int)(b_left)
                                           + Bt_right*(int)(!b_right)+ background[3*curb_right+2]*(int)(b_right) ) 
                        / (float)n_of_b ;  
    }else{
        buf2[curt*3] = target[curt*3] ;
        buf2[curt*3+1] = target[curt*3+1] ;
        buf2[curt*3+2] = target[curt*3+2] ;
    }
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
    printf("wb , hb , wt , ht , oy , ox  = %d , %d , %d , %d , %d , %d \n" , 
            wb , hb , wt , ht , oy , ox );
	
    float *fixed , *buf1, *buf2 ; 
    hipMalloc(&fixed, 3*wt*ht*sizeof(float));
    hipMalloc(&buf1, 3*wt*ht*sizeof(float));
    hipMalloc(&buf2, 3*wt*ht*sizeof(float));

    // initialize the iteration
    dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
    CalculateFixed<<<gdim, bdim>>>(  background, target, mask, fixed,
                                        wb, hb, wt, ht, oy, ox );
    hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
    hipMemcpy(buf2, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);

    // iterate
    for (int i = 0; i < 1000; ++i) {
        PoissonImageCloningIteration<<<gdim, bdim>>>(background, fixed, mask, buf1, buf2,
                                        wb, hb, wt, ht, oy, ox );
        PoissonImageCloningIteration<<<gdim, bdim>>>(background, fixed, mask, buf2, buf1,
                                        wb, hb, wt, ht, oy, ox );
    }


    hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);
    hipFree(fixed);
    hipFree(buf1);
    hipFree(buf2);
}
