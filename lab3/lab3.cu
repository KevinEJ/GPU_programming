#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>
#include <cassert>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void DownSampling_2 ( const float *a , float *b , const int wt , const int ht , const int Scale);
__global__ void UpSampling_2 ( const float *a , float *b , const int wt ,const int ht  , const int Scale);
__global__ void Initial_solution( const float *background , float *buf1 , 
                        const int wb , const int hb , const int wt , const int ht , const int oy , const int ox  ) ;

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *gradient,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox )
{	
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
    
    const int up_n    = wt*(yt-1) + xt   ; 
    const int down_n  = wt*(yt+1) + xt   ; 
    const int left_n  = wt*yt     + xt-1 ; 
    const int right_n = wt*yt     + xt+1 ; 
    
    //if( mask[up_n] < 127.0f or mask[down_n] < 127.0f )
        //bool b_up    = ( yt!=0  and mask[curt] > 127.0f and mask[up_n] > 127.0f)? false : true ; 
        //bool b_down  = ( yt!=(ht-1) and mask[curt] > 127.0f and mask[down_n] > 127.0f)? false : true ; 
        //bool b_left  = ( xt!=0  and mask[curt] > 127.0f and mask[left_n] > 127.0f)? false : true ; 
        //bool b_right = ( xt!=(wt-1) and mask[curt] > 127.0f and mask[right_n] >127.0f)? false : true ; 
        bool b_up    = ( yt==0     )? true:false   ; 
        bool b_down  = ( yt==(ht-1))? true:false   ; 
        bool b_left  = ( xt==0     )? true:false   ; 
        bool b_right = ( xt==(wt-1))? true:false   ; 
    
        //const int yb = oy+yt, xb = ox+xt;
        //const int curb_up    = wb*(yb-1) +xb  ;
        //const int curb_down  = wb*(yb+1) +xb  ;
        //const int curb_left  = wb*yb     +xb-1;
        //const int curb_right = wb*yb     +xb+1;
    
    float Rt_up    = (b_up    )? 0 :target[up_n*3]     ; 
    float Rt_down  = (b_down  )? 0 :target[down_n*3]   ; 
    float Rt_left  = (b_left  )? 0 :target[left_n*3]   ; 
    float Rt_right = (b_right )? 0 :target[right_n*3]  ; 
    float Gt_up    = (b_up    )? 0 :target[up_n*3+1]   ; 
    float Gt_down  = (b_down  )? 0 :target[down_n*3+1] ; 
    float Gt_left  = (b_left  )? 0 :target[left_n*3+1] ; 
    float Gt_right = (b_right )? 0 :target[right_n*3+1]; 
    float Bt_up    = (b_up    )? 0 :target[up_n*3+2]   ; 
    float Bt_down  = (b_down  )? 0 :target[down_n*3+2] ; 
    float Bt_left  = (b_left  )? 0 :target[left_n*3+2] ; 
    float Bt_right = (b_right )? 0 :target[right_n*3+2]; 
        
    int n_of_b =   4 -( (int)b_up + (int)b_down + (int)b_left + (int)b_right ) ;
    //if( yt < ht and xt < wt and mask[curt]>127.0f){
    if( yt >= 0 and xt>= 0 and yt < ht and xt < wt ){
        gradient[curt*3]   = ( n_of_b * target[curt*3]   - Rt_up - Rt_down - Rt_left - Rt_right) / n_of_b ;  
        gradient[curt*3+1] = ( n_of_b * target[curt*3+1] - Gt_up - Gt_down - Gt_left - Gt_right) / n_of_b ;  
        gradient[curt*3+2] = ( n_of_b * target[curt*3+2] - Bt_up - Bt_down - Bt_left - Bt_right) / n_of_b ;  
    //mask boundary condition
       /* 
        gradient[curt*3] += background[curb_up*3]*(int)(b_up) ;
        gradient[curt*3] += background[curb_down*3]*(int)(b_down) ;
        gradient[curt*3] += background[curb_left*3]*(int)(b_left) ;
        gradient[curt*3] += background[curb_right*3]*(int)(b_right) ;
        gradient[curt*3+1] += background[curb_up*3+1]*(int)(b_up) ;
        gradient[curt*3+1] += background[curb_down*3+1]*(int)(b_down) ;
        gradient[curt*3+1] += background[curb_left*3+1]*(int)(b_left) ;
        gradient[curt*3+1] += background[curb_right*3+1]*(int)(b_right) ;
        gradient[curt*3+2] += background[curb_up*3+2]*(int)(b_up) ;
        gradient[curt*3+2] += background[curb_down*3+2]*(int)(b_down) ;
        gradient[curt*3+2] += background[curb_left*3+2]*(int)(b_left) ;
        gradient[curt*3+2] += background[curb_right*3+2]*(int)(b_right) ;*/
    }else{
        //gradient[curt*3] =  128;// target[curt*3] ;
        //gradient[curt*3+1] = 0;//target[curt*3+1] ;
        //gradient[curt*3+2] = 0;//target[curt*3+2] ;
    }
}

__global__ void PoissonImageCloningIteration(
    const float *background,
	const float *fixed,
	const float *mask,
	const float *target,
	float *buf2,
	const int wb, const int hb, const int wt, const int ht,
    const int oy , const int ox , const float w , const int Scale)
{	
    const int yt = (blockIdx.y * blockDim.y + threadIdx.y);
	const int xt = (blockIdx.x * blockDim.x + threadIdx.x);
	const int curt = wt*yt+xt;
    
    const int up_n    = wt*(yt-1) + xt   ; 
    const int down_n  = wt*(yt+1) + xt   ; 
    const int left_n  = wt*yt     + xt-1 ; 
    const int right_n = wt*yt     + xt+1 ; 
    
    float Rt_up    = target[up_n*3] ; 
    float Rt_down  = target[down_n*3] ; 
    float Rt_left  = target[left_n*3] ; 
    float Rt_right = target[right_n*3] ; 
    float Gt_up    = target[up_n*3+1] ; 
    float Gt_down  = target[down_n*3+1] ; 
    float Gt_left  = target[left_n*3+1] ; 
    float Gt_right = target[right_n*3+1] ; 
    float Bt_up    = target[up_n*3+2] ; 
    float Bt_down  = target[down_n*3+2] ; 
    float Bt_left  = target[left_n*3+2] ; 
    float Bt_right = target[right_n*3+2] ; 
    //if( mask[up_n] < 127.0f or mask[down_n] < 127.0f )
        const int yb = oy+(yt*Scale) , xb = ox+(xt*Scale) ;
        const int curb       = wb*(yb) +xb  ;
        const int curb_up    = wb*(yb-1) +xb  ;
        const int curb_down  = wb*(yb+1) +xb  ;
        const int curb_left  = wb*yb     +xb-1;
        const int curb_right = wb*yb     +xb+1;

    if( yt>= 0 and xt >= 0 and  yt < ht and xt < wt and mask[curt]>127.0f){
    //mask boundary condition
        bool b_up    = ( yt==0  or mask[up_n]<127.0f)? true : false ; 
        bool b_down  = ( yt==(ht-1) or mask[down_n]<127.0f)? true : false ; 
        bool b_left  = ( xt==0  or mask[left_n]<127.0f)? true : false ; 
        bool b_right = ( xt==(wt-1) or mask[right_n]<127.0f)? true : false ; 
        int n_of_b = 4 ; // - ( (int)b_up + (int)b_down + (int)b_left + (int)b_right ) ;
        if(b_up == true or b_down == true or b_left == true or b_right == true ){
            buf2[curt*3] = background[curb*3+0] ;//target[curt*3] ;
            buf2[curt*3+1] = background[curb*3+1] ;
            buf2[curt*3+2] = background[curb*3+2] ;
        }
        else{
        buf2[curt*3+0] = ( 4*fixed[curt*3+0] + Rt_up*(int)(!b_up)      + background[3*curb_up]   *(int)(b_up)
                                           + Rt_down*(int)(!b_down)  + background[3*curb_down] *(int)(b_down)
                                           + Rt_left*(int)(!b_left)  + background[3*curb_left] *(int)(b_left)
                                           + Rt_right*(int)(!b_right)+ background[3*curb_right]*(int)(b_right) ) 
                        / (float)n_of_b ;  
        buf2[curt*3+1] = ( 4*fixed[curt*3+1] + Gt_up*(int)(!b_up)      + background[3*curb_up+1]   *(int)(b_up)
                                           + Gt_down*(int)(!b_down)  + background[3*curb_down+1] *(int)(b_down)
                                           + Gt_left*(int)(!b_left)  + background[3*curb_left+1] *(int)(b_left)
                                           + Gt_right*(int)(!b_right)+ background[3*curb_right+1]*(int)(b_right) ) 
                        / (float)n_of_b ;  
        buf2[curt*3+2] = ( 4*fixed[curt*3+2] + Bt_up*(int)(!b_up)      + background[3*curb_up+2]   *(int)(b_up)
                                           + Bt_down*(int)(!b_down)  + background[3*curb_down+2] *(int)(b_down)
                                           + Bt_left*(int)(!b_left)  + background[3*curb_left+2] *(int)(b_left)
                                           + Bt_right*(int)(!b_right)+ background[3*curb_right+2]*(int)(b_right) ) 
                        / (float)n_of_b ;  
        }
        // SOR??
        float a =  buf2[curt*3]*w + (1-w)*target[curt*3+0] ;
        float b =  buf2[curt*3+1]*w + (1-w)*target[curt*3+1] ;
        float c =  buf2[curt*3+2]*w + (1-w)*target[curt*3+2] ;
        if( a > 255 || a < 0 || b > 255 || b < 0 || c>255 || c<0)
        {}
        else{
        buf2[curt*3]   =  buf2[curt*3]   *w + (1-w)*target[curt*3+0]     ;
        buf2[curt*3+1] =  buf2[curt*3+1] *w + (1-w)*target[curt*3+1]     ;
        buf2[curt*3+2] =  buf2[curt*3+2] *w + (1-w)*target[curt*3+2]     ;
        }

    }else if( yt>= 0 and xt >= 0 and  yt < ht and xt < wt){
        buf2[curt*3] = background[curb*3+0] ;//target[curt*3] ;
        buf2[curt*3+1] = background[curb*3+1] ;
        buf2[curt*3+2] = background[curb*3+2] ;
    }

        
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox)
{
    printf("wb , hb , wt , ht , oy , ox  = %d , %d , %d , %d , %d , %d \n" , 
            wb , hb , wt , ht , oy , ox );
    const int Num_iter = 1000 ;	
    float *fixed , *buf1, *buf2 ; 
    hipMalloc(&fixed, 3*wt*ht*sizeof(float));
    hipMalloc(&buf1, 3*wt*ht*sizeof(float));
    hipMalloc(&buf2, 3*wt*ht*sizeof(float));

    // initialize the iteration
    dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
    CalculateFixed<<<gdim, bdim>>>(  background, target, mask, fixed,
                                        wb, hb, wt, ht, oy, ox );
    hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
    Initial_solution<<<gdim,bdim>>>( background , buf1 , wb , hb , wt , ht , oy , ox  ) ;
    hipMemcpy(buf2, buf1, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
    
    float w = 1 ; 
    //bool *isConv ;
    //hipMalloc(&isConv, 1*sizeof(bool));
    // Down Scaling
    int Scale = 4 ; 
    
    float *fixed_4 , *buf1_4, *buf2_4 , *mask_4 , *target_4; 
    hipMalloc(&fixed_4, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&buf1_4, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&buf2_4, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&mask_4, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&target_4, 3*wt/Scale*ht/Scale*sizeof(float));

    dim3 gdim_4(CeilDiv(wt/Scale,32), CeilDiv(ht/Scale,16)), bdim_4(32,16);
    DownSampling_2<<< gdim_4,bdim_4 >>> (target , target_4 , wt/Scale , ht/Scale , Scale );
    DownSampling_2<<< gdim_4,bdim_4 >>> (buf1  , buf1_4  , wt/Scale , ht/Scale , Scale );
    DownSampling_2<<< gdim_4,bdim_4 >>> (buf2  , buf2_4  , wt/Scale , ht/Scale , Scale );
    DownSampling_2<<< gdim_4,bdim_4 >>> (mask  , mask_4  , wt/Scale , ht/Scale , Scale );
    CalculateFixed<<< gdim_4,bdim_4 >>> (background, target_4, mask_4, fixed_4,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox );
    
    for (int i = 0; i < Num_iter*6/10; ++i) {
        PoissonImageCloningIteration<<<gdim_4, bdim_4>>>(background, fixed_4, mask_4, buf1_4, buf2_4,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox ,w , Scale);
        PoissonImageCloningIteration<<<gdim_4, bdim_4>>>(background, fixed_4, mask_4, buf2_4, buf1_4,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox ,w , Scale);
    }
    
    Scale = 2 ;

    float *fixed_2 , *buf1_2, *buf2_2 , *mask_2 , *target_2; 
    hipMalloc(&fixed_2, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&buf1_2, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&buf2_2, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&mask_2, 3*wt/Scale*ht/Scale*sizeof(float));
    hipMalloc(&target_2, 3*wt/Scale*ht/Scale*sizeof(float));
    
    dim3 gdim_2(CeilDiv(wt/Scale,32), CeilDiv(ht/Scale,16)), bdim_2(32,16);
    
    UpSampling_2<<< gdim_2,bdim_2 >>> (buf1_4  , buf1_2 , wt/Scale  , ht/Scale , Scale );
    UpSampling_2<<< gdim_2,bdim_2 >>> (buf2_4  , buf2_2 , wt/Scale  , ht/Scale , Scale );
    
    DownSampling_2<<< gdim_2,bdim_2 >>> (target , target_2 , wt/Scale , ht/Scale , Scale );
    //DownSampling_2<<< gdim_2,bdim_2 >>> (buf1  , buf1_2  , wt/Scale , ht/Scale , Scale );
    //DownSampling_2<<< gdim_2,bdim_2 >>> (buf2  , buf2_2  , wt/Scale , ht/Scale , Scale );
    DownSampling_2<<< gdim_2,bdim_2 >>> (mask  , mask_2  , wt/Scale , ht/Scale , Scale );
    CalculateFixed<<< gdim_2,bdim_2 >>> (background, target_2, mask_2, fixed_2,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox );
   

    for (int i = 0; i < Num_iter*3/10; ++i) {
        PoissonImageCloningIteration<<<gdim_2, bdim_2>>>(background, fixed_2, mask_2, buf1_2, buf2_2,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox ,w , Scale);
        PoissonImageCloningIteration<<<gdim_2, bdim_2>>>(background, fixed_2, mask_2, buf2_2, buf1_2,
                                        wb, hb, wt/Scale, ht/Scale, oy, ox ,w , Scale);
    }
    //UpSampling_2<<< gdim,bdim >>> (fixed_2 , fixed , wt , ht , Scale);
    UpSampling_2<<< gdim,bdim >>> (buf1_2  , buf1 , wt  , ht , Scale );
    UpSampling_2<<< gdim,bdim >>> (buf2_2  , buf2 , wt  , ht , Scale );
    //UpSampling_2<<< gdim,bdim >>> (mask_2  , mask);


    // iterate
    //float w = 3 ;
    //float Num_iter = 1000 ;
    w = 1 ;

    for (int i = 0; i < Num_iter*1/10; ++i) {
        PoissonImageCloningIteration<<<gdim, bdim>>>(background, fixed, mask, buf1, buf2,
                                        wb, hb, wt, ht, oy, ox ,w ,1 );
        PoissonImageCloningIteration<<<gdim, bdim>>>(background, fixed, mask, buf2, buf1,
                                        wb, hb, wt, ht, oy, ox ,w ,1 );
        w = 1 + ( (w-1) / 1.1 ) ;
    }


    hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	if(Num_iter != 0 )
    SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);
    else
    SimpleClone<<<dim3(CeilDiv(wt,32), CeilDiv(ht,16)), dim3(32,16)>>>(
		background, fixed, mask, output,
		wb, hb, wt, ht, oy, ox
	);
    hipFree(fixed);
    hipFree(buf1);
    hipFree(buf2);
}

__global__ void DownSampling_2 ( const float *a , float *b , const int wt , const int ht , const int Scale){
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt_a = (wt*Scale)*(yt*Scale)+(xt*Scale);
	const int curt_a_r = (wt*Scale)*(yt*Scale+1)+(xt*Scale);
	const int curt_a_d = (wt*Scale)*(yt*Scale)+(xt*Scale+1);
	const int curt_a_rd = (wt*Scale)*(yt*Scale+1)+(xt*Scale+1);
	const int curt_b = wt*(yt)+(xt);
   
    if( yt>= 0 and xt >= 0 and  yt < ht and xt < wt){
    //b[curt_b*3+0] = a[curt_a*3+0] + ; 
        b[curt_b*3] = ( a[curt_a*3] + a[curt_a_r*3] + a[curt_a_d*3] + a[curt_a_rd*3] ) / 4 ;  
        b[curt_b*3+1] = ( a[curt_a*3+1] + a[curt_a_r*3+1] + a[curt_a_d*3+1] + a[curt_a_rd*3+1] ) / 4 ;  
        b[curt_b*3+2] = ( a[curt_a*3+2] + a[curt_a_r*3+2] + a[curt_a_d*3+2] + a[curt_a_rd*3+2] ) / 4 ;  
    //b[curt_b*3+1] = a[curt_a*3+1]; 
    //b[curt_b*3+2] = a[curt_a*3+2]; 
    }
}
__global__ void UpSampling_2 ( const float *a , float *b , const int wt , const int ht, const int Scale){
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt_a = (wt/Scale)*(yt/Scale)+(xt/Scale);
	const int curt_b = (wt)*yt+xt;
    
    const int curt_a_right = (wt/Scale)*(yt/Scale)+(xt/Scale+1);
    const int curt_a_down  = (wt/Scale)*(yt/Scale+1)+(xt/Scale);
    const int curt_a_rd    = (wt/Scale)*(yt/Scale+1)+(xt/Scale+1);
  
    if( yt>= 0 and xt >= 0 and  yt < ht and xt < wt){
	if(Scale == 2 ){
        if(yt%Scale == 1 and yt != (ht-1)){
            if(xt%Scale == 1 and xt !=(wt-1)){
                b[curt_b*3] = ( a[curt_a*3] + a[curt_a_right*3] + a[curt_a_down*3] + a[curt_a_rd*3] ) / 4 ;  
                b[curt_b*3+1] = ( a[curt_a*3+1] + a[curt_a_right*3+1] + a[curt_a_down*3+1] + a[curt_a_rd*3+1] ) / 4 ;  
                b[curt_b*3+2] = ( a[curt_a*3+2] + a[curt_a_right*3+2] + a[curt_a_down*3+2] + a[curt_a_rd*3+2] ) / 4 ;  
            }else{
                b[curt_b*3] = ( a[curt_a*3]     + a[curt_a_down*3]  ) / 2 ;  
                b[curt_b*3+1] = ( a[curt_a*3+1] + a[curt_a_down*3+1] ) / 2 ;  
                b[curt_b*3+2] = ( a[curt_a*3+2] + a[curt_a_down*3+2] ) / 2 ;  
            }
        }
        else{
            if(xt%Scale == 1 and xt!=(wt-1)){
                b[curt_b*3] = ( a[curt_a*3]     + a[curt_a_right*3]   ) / 2 ;  
                b[curt_b*3+1] = ( a[curt_a*3+1] + a[curt_a_right*3+1] ) / 2 ;  
                b[curt_b*3+2] = ( a[curt_a*3+2] + a[curt_a_right*3+2] ) / 2 ;  
            }else{
                b[curt_b*3] =     a[curt_a*3]   ; 
                b[curt_b*3+1] =   a[curt_a*3+1] ;  
                b[curt_b*3+2] =   a[curt_a*3+2] ;  
            }
        
        }
    }
  } 
    //b[curt_b*3+0] = a[curt_a*3+0]; 
    //b[curt_b*3+1] = a[curt_a*3+1]; 
    //b[curt_b*3+2] = a[curt_a*3+2]; 
}
__global__ void Initial_solution( const float *background , float *buf1 , 
                        const int wb , const int hb , const int wt , const int ht , const int oy , const int ox  ) {

    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = (wt)*(yt)+(xt);
    
    const int yb = oy+(yt) , xb = ox+(xt) ;
    const int curb       = wb*(yb) +xb  ;

    const float R = background[curb*3+0] ; 
    const float G = background[curb*3+1] ; 
    const float B = background[curb*3+2] ;
    //const float Y =  0.299*R + 0.587*G + 0.114*B ; 
    const float U = -0.169*R - 0.331*G + 0.500*B ; 
    const float V =  0.500*R - 0.419*G - 0.081*B ; 

    const float Rt = buf1[curt*3+0] ; 
    const float Gt = buf1[curt*3+1] ; 
    const float Bt = buf1[curt*3+2] ;
    const float Yt =  0.299*Rt + 0.587*Gt + 0.114*Bt ; 
    //const float Ut = U ; 
    //const float Vt = V ; 

    const float Rf = Yt + 1.13983*(V) ; 
    const float Gf = Yt - 0.39465*(U) -0.58060*(V); 
    const float Bf = Yt + 2.03211*(U) ; 
    
    if( yt>= 0 and xt >= 0 and  yt < ht and xt < wt){
			buf1[curt*3+0] = Rf ;
			buf1[curt*3+1] = Gf ;
			buf1[curt*3+2] = Bf ;
             
    }

}
