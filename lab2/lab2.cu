#include "hip/hip_runtime.h"
#include "lab2.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 30;

struct Lab2VideoGenerator::Impl {
	int t = 0;
};

Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};

__global__ void gpu_Generation(uint8_t *input_gpu, int fsize , int t) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //extent the current frame
    //unsigned char *tempframe = new unsigned char[640*480];
    uint8_t *temp_frame = new uint8_t[640*480];
    
    for(int i = 0 ; i < 640*480 ; i++){
        temp_frame[i] = 128 ; 
    }
    for(int i = 0 ; i < 640 ; i++){
        for(int j = 0 ; j < 480 ; j++){
            int new_i = (i - 320)*1.1 + 320 ;
            int new_j = (j - 240)*1.1 + 240 ;
            //if(t==15){
            //    printf("at t=5 i,j = (%d,%d) => newij = (%d , %d ) \n" , i ,j , new_i , new_j);
            //}
            if( new_i>=0 && new_i < 640 && new_j >=0 && new_j < 480)
                temp_frame[new_i*480+new_j] = input_gpu[i*480+j];
        }
    }
    for(int i = 300 ; i < 340 ; i ++)
        for(int j = 220 ; j < 260 ; j++) 
            temp_frame[i*480+j] = 255 ; 

    for(int i = 0 ; i < 640*480 ; i++){
        input_gpu[i] = temp_frame[i] ; 
    }
}
void Lab2VideoGenerator::Generate(uint8_t *yuv) {
    //extent the current frame
    //unsigned char *tempframe = new unsigned char[640*480];
	//MemoryBuffer<uint8_t> a (H*W*1.5);
	//auto b = frameb.CreateSync(H*W*1.5);
    
	if (impl->t < 5){
        hipMemset(yuv, 0, W*H);
    //    hipMemset(yuv+H*W, 128, W*H/2);
    }else{
        gpu_Generation<<<1,1>>>(yuv,H*W*1.5,impl->t); 
    //    hipMemset(yuv+H*W, 128, W*H/2);
    }//for(int i = 0 ; )
    //rotation the extented frame   
    //set the center a block
    //
    //hipMemset(yuv, (impl->t)*255/NFRAME, W*H/2);
    //hipMemset(yuv+W*H/2, 255-((impl->t)*255/NFRAME), W*H/2);
	//hipMemset(yuv+W*H, 128, W*H/2);
    //printf("current t = %d \n" , impl->t);
	//if (impl->t < 10)
    //    hipMemset(yuv, 0, W*H);
	//else{
    //    cudaMemget
    //    hipMemset(yuv, yuv[3] , W*H);
	//}
    //hipMemset(yuv+W*H, 128, W*H/2);
    ++(impl->t);
}



