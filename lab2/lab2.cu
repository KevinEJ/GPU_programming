#include "hip/hip_runtime.h"
#include "lab2.h"
#include "SyncedMemory.h"
#include <iostream>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define SIN_12 0.087155
#define COS_12 0.996194
#define DEGREE *3.1415926/180
#define CENTER 153920
#define R      100
#define Y_SAT  200
#define CIRCLE_FRE  901
#define Y_INDEX  (i)*640+j
#define U_INDEX  640*480 + (i/2)*320+(j/2)
#define V_INDEX  640*600 + (i/2)*320+(j/2)
#define gpu_Y_INDEX  (blockIdx.x)*640+threadIdx.x
#define gpu_U_INDEX  640*480 + (blockIdx.x/2)*320+(threadIdx.x/2)
#define gpu_V_INDEX  640*600 + (blockIdx.x/2)*320+(threadIdx.x/2)
#define new_Y_INDEX  (new_i)*640+new_j
#define new_U_INDEX  640*480 + (new_i/2)*320+(new_j/2)
#define new_V_INDEX  640*600 + (new_i/2)*320+(new_j/2)

static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

struct Lab2VideoGenerator::Impl {
	int t = 0;
};

/*
__global__ void init_stuff(hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1337, idx, 0, &state[idx]);
}
__global__ void make_rand(hiprandState *state, float *randArray) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    randArray[idx] = hiprand_uniform(&state[idx]);
}*/


Lab2VideoGenerator::Lab2VideoGenerator(): impl(new Impl) {
}

Lab2VideoGenerator::~Lab2VideoGenerator() {}

void Lab2VideoGenerator::get_info(Lab2VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};
#if CPU
void gpu_Generation_firstframe(uint8_t *input_gpu) {
    //extent the current frame
    //unsigned char *tempframe = new unsigned char[640*480];
    //uint8_t *temp_frame = new uint8_t[640*480];
    //uint8_t *temp_2_frame = new uint8_t[640*480];
    
    //center point
    for(int i = 0 ; i < 640*480 ; i++){
        input_gpu[i] = 128 ; 
    }
    input_gpu[240*640+320] = 255 ;
    //other points
    int r = (sqrt(240*240 + 320*320)) / 3 ; 
    for(int i = 0 ; i < 50 ; i ++ ){
        int rand_deg = rand() % 360 ;
        float c = 0.9 + 0.8*(rand()%21)/20; 
        int rand_r      = c * r  ;
        //printf( " rand_r = %d \n" , rand_r  ) ;
        int x = rand_r*sin(rand_deg DEGREE) + 240; 
        int y = rand_r*cos(rand_deg DEGREE) + 320;
        //int idx = x * 640 + y ; 
        input_gpu[x * 640 + y ] = Y_SAT ;
        if( rand() % 4 == 0 ){
            input_gpu[640*480 + (x/2)*320 + (y/2) ] = 230 ;
            input_gpu[640*600 + (x/2)*320 + (y/2) ] = 20 + rand() % 210 ;
        }
        if( rand() % 4 == 1 ){
            input_gpu[640*480 + (x/2)*320 + (y/2) ] = 20 ;
            input_gpu[640*600 + (x/2)*320 + (y/2) ] = 20 + rand() % 210 ;
        }
        if( rand() % 4 == 2 ){
            input_gpu[640*480 + (x/2)*320 + (y/2) ] = 20 + rand() % 210 ;
            input_gpu[640*600 + (x/2)*320 + (y/2) ] = 230 ;
        }
        if( rand() % 4 == 3 ){
            input_gpu[640*480 + (x/2)*320 + (y/2) ] = 20 + rand() % 210 ;
            input_gpu[640*600 + (x/2)*320 + (y/2) ] = 20 ;
        }
        //input_gpu[640*480 + (x/2)*320 + (y/2) ] = 230 ;
        //input_gpu[640*600 + (x/2)*320 + (y/2) ] = 20 ;
    }
}
#else
__global__ void gpu_Generation_firstframe(uint8_t *input_gpu , int t , int* mv) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    input_gpu[idx] = 128 ;
    mv[idx*2] = 0 ; 
    mv[idx*2 + 1] = 0 ; 
    // Y
        int distance = sqrtf((blockIdx.x-240)*(blockIdx.x-240)+(threadIdx.x-320)*(threadIdx.x-320));
        if(distance >= R*0.8 && distance <= R*1.8){
  
  int* result = new int(0);
  hiprandState_t state;
  hiprand_init(idx*t*t,  0,  0,  &state);
  *result = hiprand(&state) % 10000001;
  
            //int rand_num = randArray[idx] ;
            int rand_num = *result ;
            //printf("this random = %d \n " , rand_num ) ;
            if( rand_num % CIRCLE_FRE == 0 ){ 
                input_gpu[idx] = Y_SAT ;
                input_gpu[gpu_U_INDEX] = (rand_num/CIRCLE_FRE)*167 % 255 ;  
                input_gpu[gpu_V_INDEX] = (rand_num/CIRCLE_FRE)*193 % 255 ;  
                mv[idx*2] = 1 + ((rand_num / CIRCLE_FRE )% 11 ); 
                mv[idx*2+1] = 1 + ((rand_num / CIRCLE_FRE) % 13 ); 
                
                if(rand_num/CIRCLE_FRE % 4 == 0 ){
                }else if(rand_num/CIRCLE_FRE % 4 == 1 ){
                    mv[idx*2] *= -1 ;
                }else if(rand_num/CIRCLE_FRE % 4 == 2 ){
                    mv[idx*2+1] *= -1 ;
                }else if(rand_num/CIRCLE_FRE % 4 == 3 ){
                    mv[idx*2] *= -1 ;
                    mv[idx*2+1] *= -1 ;
                }

                /*
                if( (rand_num/CIRCLE_FRE)%4 == 0 ){
                    input_gpu[640*480 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 230 ;
                    input_gpu[640*600 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 30  + 10*(rand_num % 20) ;
                }
                else if( (rand_num/CIRCLE_FRE) % 4 == 1 ){
                    input_gpu[640*480 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 20 ;
                    input_gpu[640*600 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 30 + 10*(rand_num % 20) ;
                }
                else if( (rand_num/CIRCLE_FRE) % 4 == 2 ){
                    input_gpu[640*480 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 30 + 10*(rand_num % 20) ;
                    input_gpu[640*600 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 230 ;
                }
                else if( (rand_num/CIRCLE_FRE) % 4 == 3 ){
                    input_gpu[640*480 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 30 + 10*(rand_num % 20) ;
                    input_gpu[640*600 + (blockIdx.x/2)*320 + (threadIdx.x/2) ] = 20 ;
                }*/
            }
        }
    if(idx == CENTER)
        input_gpu[idx] = 255 ;
}
#endif

#if CPU
void gpu_Generation_changeColor(uint8_t *input_gpu){
    input_gpu[240*640+320] = 255 ;
    for(int i = 0 ; i<480 ; i++ ){
        for(int j = 0 ; j < 640 ; j++){
            if( input_gpu[i*640+j] == Y_SAT ){
                //input_gpu[i*640+j] = 200 ;
                /*
                if(input_gpu[U_INDEX] == 230 && input_gpu[V_INDEX]!=230)
                    input_gpu[V_INDEX]+=10 ; 
                else if ( input_gpu[V_INDEX]==230 && input_gpu[U_INDEX]!=20 )
                    input_gpu[U_INDEX]-=10 ; 
                else if ( input_gpu[U_INDEX]==20 && input_gpu[V_INDEX]!=20 )
                    input_gpu[V_INDEX]-=10 ; 
                else if ( input_gpu[V_INDEX]==20 && input_gpu[U_INDEX]!=230 )
                    input_gpu[U_INDEX]+=10 ; 
                  */  
                //input_gpu[640*480 + (i/2)*640+(j/2)] = 240 ;
                //input_gpu[640*600 + (i/2)*640+(j/2)] = 10 ;
            }
        }
    }
}
#else
__global__ void gpu_Generation_changeColor(uint8_t *input_gpu , int* mv){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(input_gpu[idx] == Y_SAT /*|| input_gpu[idx] == Y_SAT+1*/ ){
            //hiprandState_t state;
            //hiprand_init( input_gpu[gpu_U_INDEX],  0,  0,  &state);
            //int result = (int)hiprand(&state) % 10000001;
            int move_u = mv[idx*2] ;
            int move_v = mv[idx*2+1] ;
           /* 
            if(result % 4 == 0 ){
            }else if(result % 4 == 1 ){
                move_u *= -1 ;
            }else if(result % 4 == 2 ){
                move_v *= -1 ;
            }else if(result % 4 == 3 ){
                move_u *= -1 ;
                move_v *= -1 ;
            }*/
             
            if( move_u + input_gpu[gpu_U_INDEX] > 255 )
                mv[idx*2] *= -1 ; 
            else if( move_u + input_gpu[gpu_U_INDEX] < 0 ) 
                mv[idx*2] *= -1 ;
 
            if( move_v + input_gpu[gpu_V_INDEX] > 255 ) 
                mv[idx*2+1] *= -1 ; 
            else if( move_v + input_gpu[gpu_V_INDEX] < 0 ) 
                mv[idx*2+1] *= -1 ; 
            
                input_gpu[gpu_U_INDEX]+=mv[idx*2] ; 
                input_gpu[gpu_V_INDEX]+=mv[idx*2+1] ; 
            /*
            if(input_gpu[gpu_U_INDEX] == 230 && input_gpu[gpu_V_INDEX]!=230)
                input_gpu[gpu_V_INDEX]+=10 ; 
            else if ( input_gpu[gpu_V_INDEX]==230 && input_gpu[gpu_U_INDEX]!=20 )
                input_gpu[gpu_U_INDEX]-=10 ; 
            else if ( input_gpu[gpu_U_INDEX]==20 && input_gpu[gpu_V_INDEX]!=20 )
                input_gpu[gpu_V_INDEX]-=10 ; 
            else if ( input_gpu[gpu_V_INDEX]==20 && input_gpu[gpu_U_INDEX]!=230 )
                input_gpu[gpu_U_INDEX]+=10 ; 
            */   
    }
    if(idx == CENTER)
        input_gpu[idx] = 255 ;
}
#endif
#if CPU 
void gpu_rotation(uint8_t *input_gpu , uint8_t *temp_frame ){
    input_gpu[240*640+320] = 255 ;
    for(int i = 0 ; i < 640 * 480 *1.5 ; i ++)
        temp_frame[i] = 128 ; 
    for(int i = 0 ; i < 480 ; i++){
        for(int j = 0 ; j < 640 ; j++){
            if(input_gpu[i*640+j] == 200){
                int new_i = (i - 240)*COS_12 - (j-320)*SIN_12 + 240 ;
                int new_j = (i - 240)*SIN_12 + (j-320)*COS_12 + 320 ;
                if( new_i>=0 && new_i < 480 && new_j >=0 && new_j < 640 ){
                    temp_frame[new_i*640+new_j] = input_gpu[i*640+j];
                    //temp_frame[640*480 +  new_i/2*320+new_j/2] = input_gpu[640*480 + i/2*320+j/2];
                    //temp_frame[640*600 +  new_i/2*320+new_j/2] = input_gpu[640*600 + i/2*320+j/2];
                    temp_frame[ new_U_INDEX] = input_gpu[ new_U_INDEX ];
                    temp_frame[ new_V_INDEX] = input_gpu[ new_V_INDEX ];
                }
            }    
        }
    }
    for(int i = 0 ; i < 640 * 480 *1.5 ; i ++)
        input_gpu[i] = temp_frame[i];
}
#else
__global__ void gpu_rotation(uint8_t *input_gpu , uint8_t *temp_frame , int direction , int* mv , int* temp_mv){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //temp_frame[idx] = 128 ; 
    //temp_frame[gpu_U_INDEX] = 128 ;
    //temp_frame[gpu_V_INDEX] = 128 ;
    if(input_gpu[idx]==Y_SAT){
        int x = blockIdx.x-240 ,  y = threadIdx.x-320 ;
        int new_i = x*COS_12 - direction*y*SIN_12 + 240 ; 
        int new_j = x*SIN_12*direction + y*COS_12 + 320 ; 
        if( new_i>=0 && new_i < 480 && new_j >=0 && new_j < 640 ){
            temp_frame[ new_i*640+new_j ] = input_gpu[idx];
            temp_frame[ new_U_INDEX ] = input_gpu[ gpu_U_INDEX ];
            temp_frame[ new_V_INDEX ] = input_gpu[ gpu_V_INDEX ];
            temp_mv[(new_i* 640 + new_j)*2 ] = mv[idx*2];
            temp_mv[(new_i* 640 + new_j)*2+1 ] = mv[idx*2+1];
        }
    }
}
__global__ void gpu_tran(uint8_t *input_gpu , uint8_t *temp_frame , int* mv , int* temp_mv){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    input_gpu[idx] = temp_frame[idx];
    input_gpu[ gpu_U_INDEX ] = temp_frame[ gpu_U_INDEX ] ;
    input_gpu[ gpu_V_INDEX ] = temp_frame[ gpu_V_INDEX ] ; 
    if( temp_frame[gpu_U_INDEX] != 128 && temp_frame[idx]!=Y_SAT )
        input_gpu[idx] = Y_SAT + 1 ;
    mv[idx*2] = temp_mv[idx*2];
    mv[idx*2+1] = temp_mv[idx*2+1];
    if(idx == CENTER)
        input_gpu[idx] = 255 ;
}
#endif

__global__ void gpu_extention(uint8_t *input_gpu , uint8_t *temp_frame , int *mv , int *temp_mv){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //int radius = 10 ; 
    //temp_frame[idx] = 128 ;//input_gpu[idx] ; 
    //temp_frame[gpu_U_INDEX] = 128;//input_gpu[gpu_U_INDEX] ;
    //temp_frame[gpu_V_INDEX] = 128;//input_gpu[gpu_V_INDEX] ;
    if(input_gpu[idx]==Y_SAT){
        for(int i = -10 ; i<=10 ; i++ ){
            for(int j = -10 ; j <= 10 ; j ++){
                if( sqrtf(i*i+j*j) < 10){
                    temp_frame[(blockIdx.x-i)*640 + (threadIdx.x-j)] = Y_SAT+1 ; 
                    temp_frame[640*480 + (blockIdx.x-i)/2*320 + (threadIdx.x-j)/2 ] = input_gpu[gpu_U_INDEX];
                    temp_frame[640*600 + (blockIdx.x-i)/2*320 + (threadIdx.x-j)/2 ] = input_gpu[gpu_V_INDEX];
                    temp_mv[  ((blockIdx.x-i)*640 + (threadIdx.x-j) )*2] = mv[idx*2] ; 
                    temp_mv[  ((blockIdx.x-i)*640 + (threadIdx.x-j) )*2 +1] = mv[idx*2+1] ;
                }
            }
        }
        temp_frame[idx] = input_gpu[idx] ; 
        temp_mv[idx*2] = mv[idx*2] ; 
        temp_mv[idx*2+1] = mv[idx*2+1] ; 
        //temp_frame[gpu_U_INDEX] = input_gpu[gpu_U_INDEX] ;
        //temp_frame[gpu_V_INDEX] = input_gpu[gpu_V_INDEX] ;
    } 
}


void Lab2VideoGenerator::Generate(uint8_t *yuv , uint8_t * temp_frame , int* mv , int* temp_mv) {
    //extent the current frame
    

    #if CPU
    uint8_t *temp_frame = new uint8_t[640*720];
    uint8_t *temp_2_frame = new uint8_t[640*480];
	#else
    hipMemset(temp_frame , 128 , W*H*1.5);
    hipMemset(temp_mv , 0 , W*H*2);

    #endif
    //printf(" impl->t = %d \n " , impl->t ); 
	//if (impl->t < 5){
        //hipMemset(yuv, 128, W*H);
        //hipMemset(yuv+H*W, 128, W*H/2);
    //}else{
        //printf(t)
    //    gpu_Generation<<<1,1>>>(yuv,temp_frame.get_gpu_rw() ); 
    //    hipMemset(yuv+H*W, 128, W*H/2);
    //}
    
	if (impl->t < 5){
        //printf( "starting:  t = %d \n" , impl->t );
        #if CPU
        for(int i = 0 ; i < 640*480*1.5 ; i++){
            yuv[i] = 128 ;
        }
        #else
            hipMemset(yuv, 128, W*H*1.5);
        #endif
        
    }
    else if (impl-> t == 5  ){
        //printf( "firstframe:  t = %d \n" , impl->t );
        #if CPU
            gpu_Generation_firstframe(yuv,temp_frame,temp_2_frame,impl->t , mv);
        #else
            hipMemset(yuv+W*H, 128, W*H/2);
            hipMemset(temp_mv , 0 , W*H*2);
            gpu_Generation_firstframe<<<480,640>>>(yuv , impl->t , mv);
            gpu_extention<<<480,640>>>(yuv , temp_frame , mv , temp_mv);
            gpu_tran<<<480,640>>>    (yuv , temp_frame , mv , temp_mv);
        #endif
    }
    else if (impl->t > 180 || (impl-> t > 5 && impl->t < 100 )){ 
        //printf( "change color:  t = %d \n" , impl->t );
        #if CPU
            gpu_Generation_changeColor(yuv);
        #else
            gpu_Generation_changeColor<<<480,640>>>(yuv , mv );
            //hipMemset(temp_mv , 0 , W*H*2);
            hipMemset(temp_mv , 0 , W*H*2);
            gpu_extention<<<480,640>>>(yuv , temp_frame , mv , temp_mv);
            gpu_tran<<<480,640>>>    (yuv , temp_frame , mv , temp_mv);
        #endif
    }
    else{ 
        //gpu_Generation_changeColor(yuv);
        //printf( "rotation :  t = %d \n" , impl->t );
        #if CPU
            gpu_Generation_changeColor(yuv);
            gpu_rotation(yuv , temp_frame);
        #else
            gpu_Generation_changeColor<<<480,640>>>(yuv , mv);
            
            hipMemset(temp_frame , 128 , W*H);
            if (impl->t / 40 % 2 == 1 ){
                printf("do right rotate \n ") ;
                gpu_rotation<<<480,640>>>(yuv , temp_frame , -1 , mv , temp_mv);
            }else{ 
                printf("do left rotate \n ") ;
                gpu_rotation<<<480,640>>>(yuv , temp_frame , 1 , mv , temp_mv);
            }
            gpu_tran<<<480,640>>>    (yuv , temp_frame , mv , temp_mv);
            //hipMemset(temp_frame.get_gpu_wo() , 128 , W*H);
            gpu_extention<<<480,640>>>(yuv , temp_frame , mv , temp_mv);
            //gpu_extention<<<480,640>>>(temp_frame.get_gpu_rw() , yuv);
            gpu_tran<<<480,640>>>    (yuv , temp_frame , mv , temp_mv);
        #endif
    }
    //rotation the extented frame   
    ++(impl->t);
}



