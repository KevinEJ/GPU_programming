#include <cstdio>
#include <cstdint>
#include <cstdlib>
#include "SyncedMemory.h"
#include "lab2.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

int main(int argc, char **argv)
{
	Lab2VideoGenerator g;
	Lab2VideoInfo i;

	g.get_info(i);
	if (i.w == 0 or i.h == 0 or i.n_frame == 0 or i.fps_n == 0 or i.fps_d == 0) {
		puts("Cannot be zero");
		abort();
	} else if (i.w%2 != 0 or i.h%2 != 0) {
		puts("Only even frame size is supported");
		abort();
	}
	unsigned FRAME_SIZE = i.w*i.h*3/2;
	//unsigned FRAME_SIZE = i.w*i.h;
    printf(" i.w = %d , i.h = %d , i,n_frame = %d , i.fps_n = %d , i.fps_d = %d \n" 
              , i.w,i.h,i.n_frame,i.fps_n,i.fps_d);
	MemoryBuffer<uint8_t> frameb(FRAME_SIZE);
	auto frames = frameb.CreateSync(FRAME_SIZE);
	FILE *fp = fopen("result.y4m", "wb");
	fprintf(fp, "YUV4MPEG2 W%d H%d F%d:%d Ip A1:1 C420\n", i.w, i.h, i.fps_n, i.fps_d);

    MemoryBuffer<uint8_t> temp_fr (FRAME_SIZE);
	auto temp_frame = temp_fr.CreateSync(FRAME_SIZE);
    MemoryBuffer<int> mv_b (640*480*2);
	auto mv = mv_b.CreateSync(640*480*2);
    MemoryBuffer<int> mv_tb (640*480*2);
	auto temp_mv = mv_tb.CreateSync(640*480*2);

	for (unsigned j = 0; j < i.n_frame; ++j) {
	    printf("j = %d \n" , j );	
        fputs("FRAME\n", fp);
		#if CPU
        g.Generate(frames.get_cpu_wo());
		#else
        g.Generate(frames.get_gpu_wo() , temp_frame.get_gpu_rw() , mv.get_gpu_rw() , temp_mv.get_gpu_rw());
		#endif
        fwrite(frames.get_cpu_ro(), sizeof(uint8_t), FRAME_SIZE, fp);
	}

	fclose(fp);
	return 0;
}
